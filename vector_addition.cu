
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib> // for rand()

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << vector[i] << " ";
    }
    std::cout << std::endl;
}

int main() {
    int N;
    std::cout << "Enter the size of the vectors: ";
    std::cin >> N;

    int* A, * B, * C;
    A = new int[N];
    B = new int[N];
    C = new int[N];

    std::cout << "Enter values for vector A:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cin >> A[i];
    }

    std::cout << "Enter values for vector B:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cin >> B[i];
    }

    int* X, * Y, * Z;
    hipMalloc(&X, N * sizeof(int));
    hipMalloc(&Y, N * sizeof(int));
    hipMalloc(&Z, N * sizeof(int));

    hipMemcpy(X, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(Y, B, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}

